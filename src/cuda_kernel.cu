#include "hip/hip_runtime.h"
__global__ void decipher(unsigned int v0, unsigned int v1, int sum, unsigned int *key, int delta, int mask)
{
    // declare shared variables inside the block
    __shared__ unsigned int v0_s = v0;
    __shared__ unsigned int v1_s = v1;
    __shared__ int sum_s = sum;

    // the number of the tread is not relevant

    v1_s = (v1 - (((v0<<4 ^ v0>>5) + v0) ^ (sum + key[sum>>11 & 3]))) & mask;
    sum_s = (sum - delta) & mask;
    v0_s = (v0 - (((v1<<4 ^ v1>>5) + v1) ^ (sum + key[sum & 3]))) & mask;

}